#include "hip/hip_runtime.h"
__global__ void matrix(float *MatrixA, float *matrixB, float *matrixC){
    int nRow = blockIdx.y * blockDim.y+  threadIdx.y;
    int nCol = blockIdx.x * blockDim.x + threadIdx.x;
    float Cval = 0.0;

    for(int i = 0; i < k; i++)
    {
        fCval += MatrixA[nRow*K+i] * MatrixB[i*n+nCol];
    }
    matrixC[nRow*n+nCol] = fCval;
}